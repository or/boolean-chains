#include <cstdint>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// --- CUDA Kernel and Device Functions ---

// Using constants directly in the device code
#define N 12
#define SIZE (1 << (N - 1))
#define MAX_LENGTH 18
#define TAUTOLOGY ((1 << N) - 1)

// Targets are moved to device constant memory for faster access
__constant__ uint32_t d_TARGETS[7];

// A simple structure to hold the state of each thread's search
struct ThreadState {
    uint32_t chain[MAX_LENGTH];
    uint32_t expressions[1000];
    uint32_t expressions_size[MAX_LENGTH];
    uint8_t unseen[SIZE];
    uint32_t choices[MAX_LENGTH];
    uint32_t num_unfulfilled_targets;
    uint32_t total_chains;
};

// Device function to check if a value is a target
__device__ inline bool is_target(uint32_t val, uint32_t &num_unfulfilled) {
    for (int i = 0; i < 7; ++i) {
        if (val == d_TARGETS[i]) {
            // This is a simplified logic. A more robust implementation
            // would handle unique target fulfillment.
            num_unfulfilled--;
            return true;
        }
    }
    return false;
}

// The core search logic encapsulated in a CUDA kernel
__global__ void search_kernel(uint32_t *global_total_chains,
                              uint32_t start_depth) {
    // Get unique thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread gets its own state
    ThreadState s;

    // --- Initialization ---
    s.total_chains = 0;
    s.num_unfulfilled_targets = 7; // NUM_TARGETS

    // Initialize unseen array
    for (int i = 0; i < SIZE; ++i) {
        s.unseen[i] = 1;
    }

    // Initial chain elements
    s.chain[0] = 0b0000000011111111 >> (16 - N);
    s.chain[1] = 0b0000111100001111 >> (16 - N);
    s.chain[2] = 0b0011001100110011 >> (16 - N);
    s.chain[3] = 0b0101010101010101 >> (16 - N);

    for (int i = 0; i < 4; i++) {
        s.unseen[s.chain[i]] = 0;
        is_target(s.chain[i], s.num_unfulfilled_targets);
    }

    s.expressions_size[3] = 0;
    for (uint32_t k = 1; k < 4; k++) {
        const uint32_t h = s.chain[k];
        const uint32_t not_h = ~h;
        for (uint32_t j = 0; j < k; j++) {
            const uint32_t g = s.chain[j];
            const uint32_t not_g = ~g;

            // Simplified ADD_EXPRESSION
            s.expressions[s.expressions_size[3]++] = g & h;
            s.unseen[g & h] = 0;
            s.expressions[s.expressions_size[3]++] = g | h;
            s.unseen[g | h] = 0;
            s.expressions[s.expressions_size[3]++] = g ^ h;
            s.unseen[g ^ h] = 0;
            s.expressions[s.expressions_size[3]++] = not_g & h;
            s.unseen[not_g & h] = 0;
            s.expressions[s.expressions_size[3]++] = g & not_h;
            s.unseen[g & not_h] = 0;
        }
    }

    // --- Main Search Loop ---
    // This is a simplified, non-recursive version of the original loop
    // structure. A full translation of the goto-based state machine is complex
    // for a direct GPU port. This iterative approach demonstrates the basic
    // principle.
    uint32_t chain_size = 4;
    s.choices[chain_size] =
        tid; // Each thread starts at a different initial choice

    while (chain_size < MAX_LENGTH) {
        if (s.choices[chain_size] >= s.expressions_size[chain_size - 1]) {
            // Backtrack
            chain_size--;
            if (chain_size <= start_depth)
                break; // Stop if we backtrack too far

            // Restore unseen state for the expressions of the level we are
            // leaving
            for (uint32_t i = 0; i < s.expressions_size[chain_size]; ++i) {
                s.unseen[s.expressions[i]] = 1;
            }

            s.choices[chain_size]++;
            continue;
        }

        uint32_t expr_idx = s.choices[chain_size];
        s.chain[chain_size] = s.expressions[expr_idx];

        if (s.unseen[s.chain[chain_size]] == 0) {
            s.choices[chain_size]++;
            continue;
        }

        s.total_chains++;

        // Check for solution
        if (is_target(s.chain[chain_size], s.num_unfulfilled_targets) &&
            s.num_unfulfilled_targets == 0) {
            // In a real scenario, we would save the solution chain here.
            // For now, we just count it.
            // To print, we'd need to copy the chain to a global buffer.
            // printf is very slow on GPUs and should be used for debugging
            // only.
        }

        // Move to next level
        uint32_t prev_chain_size = chain_size;
        chain_size++;
        s.choices[chain_size] = 0;
        s.expressions_size[chain_size - 1] =
            s.expressions_size[prev_chain_size - 1];

        // Generate new expressions
        const uint32_t h = s.chain[prev_chain_size];
        const uint32_t not_h = ~h;
        for (uint32_t j = 0; j < prev_chain_size; j++) {
            const uint32_t g = s.chain[j];
            const uint32_t not_g = ~s.chain[j];
            s.expressions[s.expressions_size[chain_size - 1]++] = g & h;
            s.expressions[s.expressions_size[chain_size - 1]++] = g & not_h;
            s.expressions[s.expressions_size[chain_size - 1]++] = not_g & h;
            s.expressions[s.expressions_size[chain_size - 1]++] = g | h;
            s.expressions[s.expressions_size[chain_size - 1]++] = g ^ h;
        }
    }

    // Atomically add this thread's count to the global total
    atomicAdd(global_total_chains, s.total_chains);
}

// --- Host Code ---

void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA Error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char **argv) {
    printf("Starting CUDA computation...\n");

    // --- Device Setup ---
    int deviceId;
    hipDeviceProp_t props;
    hipGetDevice(&deviceId);
    checkCUDAError("hipChooseDevice");
    hipGetDeviceProperties(&props, deviceId);
    checkCUDAError("hipGetDeviceProperties");
    printf("Using device: %s\n", props.name);
    printf("Compute Capability: %d.%d\n", props.major, props.minor);

    // --- Data Initialization ---
    const uint32_t h_TARGETS[] = {
        ((~(uint32_t)0b1011011111100011) >> (16 - N)) & TAUTOLOGY,
        ((~(uint32_t)0b1111100111100100) >> (16 - N)) & TAUTOLOGY,
        ((~(uint32_t)0b1101111111110100) >> (16 - N)) & TAUTOLOGY,
        ((~(uint32_t)0b1011011011011110) >> (16 - N)) & TAUTOLOGY,
        ((~(uint32_t)0b1010001010111111) >> (16 - N)) & TAUTOLOGY,
        ((~(uint32_t)0b1000111111110011) >> (16 - N)) & TAUTOLOGY,
        (((uint32_t)0b0011111011111111) >> (16 - N)) & TAUTOLOGY,
    };

    // Copy targets to device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_TARGETS), h_TARGETS, sizeof(h_TARGETS));
    checkCUDAError("hipMemcpyToSymbol");

    // Allocate memory on the GPU for the global chain counter
    uint32_t *d_total_chains;
    hipMalloc(&d_total_chains, sizeof(uint32_t));
    checkCUDAError("hipMalloc");
    hipMemset(d_total_chains, 0, sizeof(uint32_t));
    checkCUDAError("hipMemset");

    // --- Kernel Launch ---
    // Configure the grid and block dimensions.
    // These values may need tuning based on the GPU.
    int num_blocks = 256;
    int threads_per_block = 256;
    dim3 dimGrid(num_blocks);
    dim3 dimBlock(threads_per_block);

    printf("Launching kernel with %d blocks and %d threads per block.\n",
           num_blocks, threads_per_block);

    // Launch the kernel
    search_kernel<<<dimGrid, dimBlock>>>(d_total_chains, 4);
    checkCUDAError("Kernel launch");

    // Synchronize to wait for the kernel to finish
    hipDeviceSynchronize();
    checkCUDAError("hipDeviceSynchronize");

    // --- Retrieve Results ---
    uint32_t h_total_chains = 0;
    hipMemcpy(&h_total_chains, d_total_chains, sizeof(uint32_t),
               hipMemcpyDeviceToHost);
    checkCUDAError("hipMemcpy from device");

    printf("Computation finished.\n");
    printf("Total chains found (approximate): %llu\n", h_total_chains);

    // --- Cleanup ---
    hipFree(d_total_chains);
    checkCUDAError("hipFree");

    hipDeviceReset();
    checkCUDAError("hipDeviceReset");

    return 0;
}
