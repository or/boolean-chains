#include <algorithm>
#include <bitset>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <random>
#include <stdio.h>
#include <vector>

// --- Problem Constants ---
// These constants define the parameters of the search problem.

// N defines the number of bits in the boolean functions.
#define N 12
// The maximum length of the logic chain to search for.
#define MAX_LENGTH 18
// The total number of possible functions is 2^(N-1), as f(x) = f(~x).
#define SIZE (1 << (N - 1))
// A bitmask representing a tautology (all ones).
#define TAUTOLOGY ((1 << N) - 1)
// The number of target functions we are trying to synthesize.
#define NUM_TARGETS 7

#define MAX_SOLUTIONS 1000

// The maximum number of parallel search tasks to generate on the CPU.
// Each task will be assigned to a CUDA thread.
#define MAX_GPU_TASKS 65536

// A structure to store a chain on the GPU.
struct Chain {
  uint32_t length;
  uint32_t expressions[1000];
  uint32_t expressions_size[MAX_LENGTH];
  uint32_t chain[MAX_LENGTH];
  uint32_t choices[MAX_LENGTH];
  uint8_t unseen[SIZE];
};

// __constant__ memory is fast, read-only memory accessible by all threads.
// It's perfect for storing the target functions.
__constant__ uint32_t d_TARGETS[NUM_TARGETS];

// A device-side lookup table to quickly check if a function is a target.
// Initialized once from the host.
__device__ uint8_t d_TARGET_LOOKUP[SIZE];

#define ADD_EXPRESSION(value, chain, chain_size)                               \
  chain.expressions[chain.expressions_size[chain_size]] = value;               \
  chain.expressions_size[chain_size] += chain.unseen[value];                   \
  chain.unseen[value] = 0;

#define ADD_EXPRESSION_TARGET(value, chain, chain_size)                        \
  {                                                                            \
    const uint32_t v = value;                                                  \
    const uint32_t a = chain.unseen[v] & d_TARGET_LOOKUP[v];                   \
    chain.expressions[chain.expressions_size[chain_size]] = v;                 \
    chain.expressions_size[chain_size] += a;                                   \
    chain.unseen[v] &= ~a;                                                     \
  }

#define GENERATE_NEW_EXPRESSIONS(chain_size, chain_struct, add_expression)     \
  {                                                                            \
    chain_struct.expressions_size[chain_size] =                                \
        chain_struct.expressions_size[chain_size - 1];                         \
    const uint32_t h = chain_struct.chain[chain_size - 1];                     \
    const uint32_t not_h = ~h;                                                 \
                                                                               \
    for (int j = 0; j < chain_size - 1; j++) {                                 \
      const uint32_t g = chain_struct.chain[j];                                \
      const uint32_t not_g = ~chain_struct.chain[j];                           \
      add_expression(g & h, chain_struct, chain_size);                         \
      add_expression(not_g & h, chain_struct, chain_size);                     \
      add_expression(g & not_h, chain_struct, chain_size);                     \
      add_expression(g ^ h, chain_struct, chain_size);                         \
      add_expression(g | h, chain_struct, chain_size);                         \
    }                                                                          \
  }

#define LOOP(CS, PREV_CS, NEXT_CS, chain_struct)                               \
  loop_##CS : if (CS < MAX_LENGTH) {                                           \
    if (chain_struct.choices[CS] < chain_struct.expressions_size[CS]) {        \
      chain_struct.chain[CS] =                                                 \
          chain_struct.expressions[chain_struct.choices[CS]];                  \
                                                                               \
      if (CS < MAX_LENGTH - 1 && NEXT_CS >= MAX_LENGTH - NUM_TARGETS) {        \
        if (__builtin_expect(                                                  \
                NEXT_CS + num_unfulfilled_targets -                            \
                        d_TARGET_LOOKUP[chain_struct.chain[CS]] ==             \
                    MAX_LENGTH,                                                \
                1)) {                                                          \
          tmp_chain_size = NEXT_CS;                                            \
          generated_chain_size = CS;                                           \
          tmp_num_unfulfilled_targets =                                        \
              num_unfulfilled_targets -                                        \
              d_TARGET_LOOKUP[chain_struct.chain[CS]];                         \
          uint32_t j = chain_struct.choices[CS] + 1;                           \
                                                                               \
          next_##CS : if (__builtin_expect(tmp_chain_size < MAX_LENGTH, 1)) {  \
            GENERATE_NEW_EXPRESSIONS(tmp_chain_size, chain_struct,             \
                                     ADD_EXPRESSION_TARGET)                    \
            generated_chain_size = tmp_chain_size;                             \
                                                                               \
            for (; j < chain_struct.expressions_size[tmp_chain_size]; ++j) {   \
              if (__builtin_expect(                                            \
                      d_TARGET_LOOKUP[chain_struct.expressions[j]], 0)) {      \
                chain_struct.chain[tmp_chain_size] =                           \
                    chain_struct.expressions[j];                               \
                tmp_num_unfulfilled_targets--;                                 \
                tmp_chain_size++;                                              \
                if (__builtin_expect(!tmp_num_unfulfilled_targets, 0)) {       \
                  chain.length = tmp_chain_size;                               \
                  uint32_t sol_idx = atomicAdd(solution_count, 1);             \
                  if (sol_idx < MAX_SOLUTIONS) {                               \
                    solutions[sol_idx] = chain_struct;                         \
                  }                                                            \
                  break;                                                       \
                }                                                              \
                j++;                                                           \
                goto next_##CS;                                                \
              }                                                                \
            }                                                                  \
          }                                                                    \
                                                                               \
          for (uint32_t i = chain_struct.expressions_size[CS];                 \
               i < chain_struct.expressions_size[generated_chain_size]; i++) { \
            chain_struct.unseen[chain_struct.expressions[i]] = 1;              \
          }                                                                    \
                                                                               \
          chain_struct.choices[CS] +=                                          \
              1 + (d_TARGET_LOOKUP[chain_struct.chain[CS]] << 16);             \
          goto loop_##CS;                                                      \
        }                                                                      \
      }                                                                        \
                                                                               \
      num_unfulfilled_targets -= d_TARGET_LOOKUP[chain_struct.chain[CS]];      \
                                                                               \
      chain_struct.choices[NEXT_CS] = chain_struct.choices[CS] + 1;            \
      GENERATE_NEW_EXPRESSIONS(NEXT_CS, chain_struct, ADD_EXPRESSION)          \
                                                                               \
      goto loop_##NEXT_CS;                                                     \
    }                                                                          \
                                                                               \
    if (CS > 6) {                                                              \
      for (uint32_t i = chain_struct.expressions_size[PREV_CS];                \
           i < chain_struct.expressions_size[CS]; i++) {                       \
        chain_struct.unseen[chain_struct.expressions[i]] = 1;                  \
      }                                                                        \
      num_unfulfilled_targets += d_TARGET_LOOKUP[chain_struct.chain[PREV_CS]]; \
      chain_struct.choices[PREV_CS] +=                                         \
          1 + (d_TARGET_LOOKUP[chain_struct.chain[PREV_CS]] << 16);            \
      goto loop_##PREV_CS;                                                     \
    } else {                                                                   \
      continue;                                                                  \
    }                                                                          \
  }

/**
 * @brief The main CUDA kernel for finding optimal logic chains.
 * Each thread executes this kernel to explore a part of the search space.
 * The recursive search of the original C++ code is transformed into an
 * iterative search using a manually managed stack (`search_stack`).
 *
 * @param initial_chains An array of starting chains generated by the host.
 * @param num_tasks The total number of starting chains to process.
 * @param solutions A device buffer to store found solutions.
 * @param solution_count An atomic counter for the number of solutions found.
 */
__global__ void find_optimal_chain_kernel(const Chain *initial_chains,
                                          uint32_t num_tasks,
                                          uint32_t tasks_per_thread,
                                          Chain *solutions,
                                          uint32_t *solution_count) {
  // Determine which task this thread is responsible for.
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (uint32_t task_id = idx * tasks_per_thread; task_id < idx * tasks_per_thread + tasks_per_thread && task_id < num_tasks;
       task_id++) {
    Chain chain = initial_chains[task_id];
    uint8_t num_unfulfilled_targets = NUM_TARGETS;
    uint8_t tmp_num_unfulfilled_targets;
    uint8_t tmp_chain_size;
    uint8_t generated_chain_size;
    chain.choices[chain.length] = chain.choices[chain.length - 1] + 1;
    GENERATE_NEW_EXPRESSIONS(chain.length, chain, ADD_EXPRESSION)

    // Calculate how many targets are unfulfilled by this initial chain.
    for (uint32_t i = 0; i < chain.length; i++) {
      num_unfulfilled_targets -= d_TARGET_LOOKUP[chain.chain[i]];
    }

    // If the initial chain is already a solution, store it.
    if (num_unfulfilled_targets == 0) {
      uint32_t sol_idx = atomicAdd(solution_count, 1);
      if (sol_idx < MAX_SOLUTIONS) {
        solutions[sol_idx] = chain;
      }
      return;
    }

    LOOP(8, 7, 9, chain)
    LOOP(9, 8, 10, chain)
    LOOP(10, 9, 11, chain)
    LOOP(11, 10, 12, chain)
    LOOP(12, 11, 13, chain)
    LOOP(13, 12, 14, chain)
    LOOP(14, 13, 15, chain)
    LOOP(15, 14, 16, chain)
    LOOP(16, 15, 17, chain)
    LOOP(17, 16, 18, chain)
    LOOP(18, 17, 19, chain)
    LOOP(19, 18, 20, chain)

  loop_20: // shouldn't be used

  loop_7:
    continue;
  }
}

// --- Host-side C++ Code ---

// Helper to print a chain's binary representation.
void print_chain(const Chain &s, const uint8_t *target_lookup) {
  printf("chain (%d):\n", s.length);
  for (uint32_t i = 0; i < s.length; i++) {
    printf("x%d", i + 1);
    for (uint32_t j = 0; j < i; j++) {
      for (uint32_t k = j + 1; k < i; k++) {
        char op = 0;
        if (s.chain[i] == (s.chain[j] & s.chain[k])) {
          op = '&';
        } else if (s.chain[i] == (s.chain[j] | s.chain[k])) {
          op = '|';
        } else if (s.chain[i] == (s.chain[j] ^ s.chain[k])) {
          op = '^';
        } else if (s.chain[i] == ((~s.chain[j]) & s.chain[k])) {
          op = '<';
        } else if (s.chain[i] == (s.chain[j] & (~s.chain[k]))) {
          op = '>';
        } else {
          continue;
        }

        printf(" = x%d %c x%d", j + 1, op, k + 1);
      }
    }
    printf(" = %s", std::bitset<N>(s.chain[i]).to_string().c_str());
    if (target_lookup[s.chain[i]]) {
      printf(" [target]");
    }
    printf("\n");
  }
}

int main(int argc, char *argv[]) {
  printf("Starting logic chain synthesis for N=%d\n", N);
  printf("Max length: %d, Num targets: %d\n", MAX_LENGTH, NUM_TARGETS);

  // --- CUDA Device Setup ---
  uint32_t dev_id = 0;
  hipDeviceProp_t props;
  hipSetDevice(dev_id);
  hipGetDeviceProperties(&props, dev_id);
  printf("Using GPU: %s\n", props.name);
  if (props.major < 3) {
    printf("Error: This code requires a GPU with compute capability 3.0 or "
           "higher.\n");
    return 1;
  }

  // --- Host-side Data Initialization ---
  uint32_t host_targets[NUM_TARGETS] = {
      ((~(uint32_t)0b1011011111100011) >> (16 - N)) & TAUTOLOGY,
      ((~(uint32_t)0b1111100111100100) >> (16 - N)) & TAUTOLOGY,
      ((~(uint32_t)0b1101111111110100) >> (16 - N)) & TAUTOLOGY,
      ((~(uint32_t)0b1011011011011110) >> (16 - N)) & TAUTOLOGY,
      ((~(uint32_t)0b1010001010111111) >> (16 - N)) & TAUTOLOGY,
      ((~(uint32_t)0b1000111111110011) >> (16 - N)) & TAUTOLOGY,
      (((uint32_t)0b0011111011111111) >> (16 - N)) & TAUTOLOGY,
  };

  uint8_t target_lookup[SIZE] = {0};
  for (uint32_t i = 0; i < NUM_TARGETS; i++) {
    target_lookup[host_targets[i]] = 1;
  }

  uint32_t start_i = 1;
  // -c for chunk mode, only complete one slice of the depth given by the
  // progress vector
  if (argc > 1 && strcmp(argv[1], "-c") == 0) {
    start_i++;
  }

  uint32_t start_indices_size __attribute__((aligned(64))) = 0;
  uint16_t start_indices[100] __attribute__((aligned(64))) = {0};

  // --- Generate Initial Search Tasks on the CPU ---
  printf("Generating initial search tasks on the CPU...\n");
  std::vector<Chain> initial_tasks;

  Chain base_chain;
  base_chain.length = 4;
  base_chain.chain[0] = 0b0000000011111111 >> (16 - N);
  base_chain.chain[1] = 0b0000111100001111 >> (16 - N);
  base_chain.chain[2] = 0b0011001100110011 >> (16 - N);
  base_chain.chain[3] = 0b0101010101010101 >> (16 - N);

  for (uint32_t i = 0; i < base_chain.length; i++) {
    start_indices[start_indices_size++] = 0;
  }

  // read the progress vector, e.g 5 2 9, commas will be ignored: 5, 2, 9
  for (uint32_t i = start_i; i < argc; i++) {
    start_indices[start_indices_size++] = atoi(argv[i]);
  }

  memset(base_chain.unseen, 1, sizeof(base_chain.unseen));
  base_chain.unseen[0] = 0;
  for (int32_t k = 0; k < base_chain.length; k++) {
    base_chain.unseen[base_chain.chain[k]] = 0;
    base_chain.choices[k] = 0xffffffff;
  }

  memset(base_chain.expressions_size, 0, sizeof(base_chain.expressions_size));
  // only do it up to length - 1, the length step is done below
  for (int32_t k = 1; k < base_chain.length - 1; k++) {
    GENERATE_NEW_EXPRESSIONS(k + 1, base_chain, ADD_EXPRESSION);
    // printf("k: %d, expr size: %d\n", k + 1, base_chain.expressions_size[k +
    // 1]);
  }

  while (base_chain.length < start_indices_size) {
    GENERATE_NEW_EXPRESSIONS(base_chain.length, base_chain, ADD_EXPRESSION);
    base_chain.choices[base_chain.length] = start_indices[base_chain.length];
    base_chain.chain[base_chain.length] =
        base_chain.expressions[base_chain.choices[base_chain.length]];
    base_chain.length++;
  }

  // Unroll the first few levels of the search to create independent tasks
  uint32_t cpu_search_depth = 4;
  std::vector<Chain> queue;
  queue.push_back(base_chain);

  for (uint32_t depth = 0; depth < cpu_search_depth; ++depth) {
    // printf("generating at depth %d: %d in queue\n", depth, queue.size());
    std::vector<Chain> next_queue;
    for (auto &current : queue) {
      GENERATE_NEW_EXPRESSIONS(current.length, current, ADD_EXPRESSION);
      // printf("chain: %d length, %d expression size\n", current.length,
      //        current.expressions_size[current.length]);
      for (uint32_t i = current.choices[current.length - 1] + 1;
           i < current.expressions_size[current.length]; ++i) {
        current.choices[current.length] = i;
        current.chain[current.length] =
            current.expressions[current.choices[current.length]];
        current.length++;
        next_queue.push_back(current);
        current.length--;
      }
    }
    queue = next_queue;
  }
  initial_tasks = queue;
  printf("Generated %zu tasks to be processed by the GPU.\n",
         initial_tasks.size());

  std::random_device rd;
  std::default_random_engine rng(rd());

  // Shuffle the vector
  std::shuffle(initial_tasks.begin(), initial_tasks.end(), rng);

  // --- Allocate and Transfer Data to GPU ---
  hipError_t err;

  Chain *d_initial_chains;
  err = hipMalloc(&d_initial_chains, initial_tasks.size() * sizeof(Chain));
  if (err != hipSuccess) {
    printf("hipMalloc failed: %s\n", hipGetErrorString(err));
    return 1;
  }

  err =
      hipMemcpy(d_initial_chains, initial_tasks.data(),
                 initial_tasks.size() * sizeof(Chain), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("hipMemcpy failed: %s\n", hipGetErrorString(err));
    return 1;
  }

  Chain *d_solutions;
  hipMalloc(&d_solutions, MAX_SOLUTIONS * sizeof(Chain));

  uint32_t *d_solution_count;
  hipMalloc(&d_solution_count, sizeof(uint32_t));
  hipMemset(d_solution_count, 0, sizeof(uint32_t));

  // Copy data to __constant__ and __device__ global memory
  hipMemcpyToSymbol(HIP_SYMBOL(d_TARGETS), host_targets, NUM_TARGETS * sizeof(uint32_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_TARGET_LOOKUP), target_lookup, SIZE * sizeof(uint8_t));

  // --- Launch Kernel ---
  printf("Launching CUDA kernel...\n");
  uint32_t num_tasks = initial_tasks.size();
  uint32_t tasks_per_thread = num_tasks / MAX_GPU_TASKS + 1;
  uint32_t num_threads = num_tasks / tasks_per_thread + 1;
  uint32_t threads_per_block = 256;
  uint32_t blocks_per_grid =
      (num_threads + threads_per_block - 1) / threads_per_block;

  find_optimal_chain_kernel<<<blocks_per_grid, threads_per_block>>>(
      d_initial_chains, num_tasks, tasks_per_thread, d_solutions,
      d_solution_count);

  // Synchronize to wait for the kernel to finish and check for errors.
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel launch failed: %s\n", hipGetErrorString(err));
  }

  // --- Retrieve and Print Results ---
  printf("Kernel finished. Retrieving results...\n");
  int solution_count_host = 0;
  hipMemcpy(&solution_count_host, d_solution_count, sizeof(uint32_t),
             hipMemcpyDeviceToHost);
  printf("found %d solutions, only kept up to %d\n", solution_count_host,
         MAX_SOLUTIONS);
  solution_count_host = std::min(solution_count_host, MAX_SOLUTIONS);

  if (solution_count_host > 0) {
    std::vector<Chain> solutions_host(solution_count_host);
    hipMemcpy(solutions_host.data(), d_solutions,
               solution_count_host * sizeof(Chain), hipMemcpyDeviceToHost);

    for (const auto &s : solutions_host) {
      print_chain(s, target_lookup);
    }
  } else {
    printf("No solutions found.\n");
  }

  // --- Cleanup ---
  hipFree(d_initial_chains);
  hipFree(d_solutions);
  hipFree(d_solution_count);

  return 0;
}
