#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <stdio.h>

__global__ void cuda_hello(int val) { printf("Hello! %d\n", val); }

int main(int argc, char **argv) {
    int dev_id;
    hipDeviceProp_t props;

    dev_id = findCudaDevice(argc, (const char **)argv);

    checkCudaErrors(hipGetDevice(&dev_id));
    checkCudaErrors(hipGetDeviceProperties(&props, dev_id));

    dim3 dimGrid(2, 2);
    dim3 dimBlock(2, 2, 2);

    cuda_hello<<<dimGrid, dimBlock>>>(16);

    hipDeviceSynchronize();

    return 0;
}
